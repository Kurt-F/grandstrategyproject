#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <fstream>
#include <string>
#include "OpenCLContext.h"
#ifdef __APPLE__
#include "OpenCL/opencl.h"
#else
#include "CL/cl.h"
#endif

int* multiplyMatrices(int* matrx_a, int* matrix_b, int columns_a, int rows_a, int columns_b, int rows_b);
int*multiplyMatricesOpenCL(int* matrx_a, int* matrix_b, int columns_a, int rows_a, int columns_b, int rows_b);
void printMatrix(int *matrix, int columns, int rows, std::string file_name = "");

void CheckError(cl_int error)
{
	if (error != CL_SUCCESS) {
		std::cerr << "OpenCL call failed with error " << error << std::endl;
		std::exit(1);
	}
}


std::string LoadKernel(const char* name)
{
	std::ifstream in(name);
	std::string result(
		(std::istreambuf_iterator<char>(in)),
		std::istreambuf_iterator<char>());
	return result;
}

cl_program CreateProgram(const std::string& source,
	cl_context context)
{
	size_t lengths[1] = { source.size() };
	const char* sources[1] = { source.data() };

	cl_int error = 0;
	cl_program program = clCreateProgramWithSource(context, 1, sources, lengths, &error);

	return program;
}


int main()
{
	int num_columns_a;
	int num_rows_a; 
	int num_columns_b; 
	int num_rows_b; 
	int* matrix_a; 
	int* matrix_b;
	int*matrix_output;
	int openCL_or_CPP;
	std::cout << "How many columns in the first matrix?\n";
	std::cin >> num_columns_a; 
	num_rows_b = num_columns_a; 
	std::cout << "Since there must be as many rows in the second matrix as there are columns in the first, there are " << num_rows_b << " rows in the second matrix\n";
	std::cout << "How many rows in the first matrix?\n";
	std::cin >> num_rows_a; 
	std::cout << "How many columns in the second matrix?\n";
	std::cin >> num_columns_b;
	int min = std::min(std::min(num_rows_a,num_columns_a), std::min(num_rows_b, num_columns_b));
	if (min <= 0)
	{
		std::cout << "Invalid input; all values must be above 0\n";
		return -1;
	}
	matrix_a = new int[num_rows_a * num_columns_a];
	for (int i = 0; i < num_rows_a; i++)
	{
		for (int j = 0; j < num_columns_a; j++)
		{
			matrix_a[(i * num_columns_a) + j] = i + j * i + j;
		//matrix_a[(i * num_columns_a) + j] = 1;
		}
	}
	matrix_b = new int[num_rows_b * num_columns_b];
	for (int i = 0; i < num_rows_b; i++)
	{
		for (int j = 0; j < num_columns_b; j++)
		{
			matrix_b[(i * num_columns_b) + j] = j * 3 + 1 + i;
			//matrix_b[(i * num_columns_a) + j] = 1;
		}
	}
	std::cout << "Do you want to multiply these matrices using OpenCl or regular C++?(0 or 1)\n";
	std::cin >> openCL_or_CPP;
	printMatrix(matrix_a, num_columns_a, num_rows_a);
	printMatrix(matrix_b, num_columns_b, num_rows_b);
	if (openCL_or_CPP == 0)
	{
		matrix_output = multiplyMatricesOpenCL(matrix_a, matrix_b, num_columns_a, num_rows_a, num_columns_b, num_rows_b);
		printMatrix(matrix_output, num_columns_b, num_rows_a, "OPENCL.txt");
	}

	else
	{
    matrix_output = multiplyMatrices(matrix_a, matrix_b, num_columns_a, num_rows_a, num_columns_b, num_rows_b);
	printMatrix(matrix_output, num_columns_b, num_rows_a, "NOTOPENCL.txt");
	}
	return 0; 
}


int*multiplyMatricesOpenCL(int* matrix_a, int* matrix_b, int columns_a, int rows_a, int columns_b, int rows_b)
{
	size_t global_work_size[3];
	global_work_size[0] = rows_a * columns_b;
	global_work_size[1] = 0;
	global_work_size[2] = 0;
	int *matrix_out = new int[rows_a * columns_b];
	OpenCLContext context = OpenCLContext();
	cl_int error = 0;
	cl_program program = CreateProgram(LoadKernel("multiply.cl"), context.GetClContext());
	int b = clBuildProgram(program, context.getDeviceIdCount(), context.getDeviceIds().data(), nullptr, nullptr, nullptr);
	cl_kernel kernel = clCreateKernel(program, "MULT", &error);
	cl_mem buf_matrix_a = clCreateBuffer(context.GetClContext(), CL_MEM_READ_WRITE | CL_MEM_COPY_HOST_PTR, sizeof(int) * columns_a * rows_a, matrix_a, &error);
	cl_mem buf_matrix_b = clCreateBuffer(context.GetClContext(), CL_MEM_READ_WRITE | CL_MEM_COPY_HOST_PTR, sizeof(int) * columns_b * rows_b, matrix_b, &error);
	cl_mem out_buffer = clCreateBuffer(context.GetClContext(), CL_MEM_READ_WRITE | CL_MEM_USE_HOST_PTR, sizeof(int) * rows_a * columns_b, matrix_out, &error);
	const int ROWS_INPUT = rows_a;
	const int COLUMNS_INPUT = columns_b;
	CheckError(clSetKernelArg(kernel, 0, sizeof(cl_mem), &buf_matrix_a));
	CheckError(clSetKernelArg(kernel, 1, sizeof(cl_mem), &buf_matrix_b));
	CheckError(clSetKernelArg(kernel, 2, sizeof(cl_mem), &out_buffer));
	CheckError(clSetKernelArg(kernel, 3, sizeof(int), &columns_a));
	CheckError(clSetKernelArg(kernel, 4, sizeof(int), &rows_a));
	CheckError(clSetKernelArg(kernel, 5, sizeof(int), &columns_b));
	cl_command_queue queue = clCreateCommandQueue(context.GetClContext(), context.getDeviceIds().data()[0], 0, &error);
	int a = clEnqueueNDRangeKernel(queue, kernel, 1, nullptr, global_work_size, nullptr, 0, nullptr, nullptr);
	int c = clEnqueueReadBuffer(queue, out_buffer, true, 0, global_work_size[0] * sizeof(int), matrix_out, 0, nullptr, nullptr);
	return matrix_out;
}

int* multiplyMatrices(int* matrix_a, int* matrix_b, int columns_a,int rows_a, int columns_b, int rows_b)
{
	int* matrix_output;
	int columns_output, rows_output;
	int temp;
	rows_output = rows_a;
	columns_output = columns_b;
	matrix_output = new int[rows_output * columns_output];
	for (int i = 0; i < rows_output; i++)
	{
		for (int j = 0; j < columns_output; j++)
		{
			temp = 0;
			for (int k = 0; k < columns_a; k++)
			{
				int matrix_a_result = matrix_a[(columns_a * i) + k];
				int matrix_b_result =  matrix_b[(k * columns_b) + j];
				temp += matrix_a_result * matrix_b_result;
			}
			matrix_output[(i * columns_output) + j] = temp;
		}
	}
	return matrix_output;
}

void printMatrix(int *matrix, int columns, int rows, std::string file_name)
{
	if (file_name.size() != 0)
	{
		std::ofstream myfile; 
		myfile.open(file_name);
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < columns; j++)
			{
				myfile << matrix[(i * columns) + j] << " ";
			}
			myfile << "\n";
		}
		myfile.close();
	}
	if (columns > 10 || rows > 10)
	{
		return;
	}
	std::cout << "Printing a matrix\n";
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < columns; j++)
		{
			std::cout << matrix[(i * columns) + j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "Done printing a matrix\n";
}