#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <fstream>
#include "OpenCLContext.h"
#ifdef __APPLE__
#include "OpenCL/opencl.h"
#else
#include "CL/cl.h"
#endif

int* multiplyMatrices(int* matrx_a, int* matrix_b, int columns_a, int rows_a, int columns_b, int rows_b);
int*multiplyMatricesOpenCL(int* matrx_a, int* matrix_b, int columns_a, int rows_a, int columns_b, int rows_b);
void printMatrix(int *matrix, int columns, int rows);


std::string LoadKernel(const char* name)
{
	std::ifstream in(name);
	std::string result(
		(std::istreambuf_iterator<char>(in)),
		std::istreambuf_iterator<char>());
	return result;
}

cl_program CreateProgram(const std::string& source,
	cl_context context)
{
	size_t lengths[1] = { source.size() };
	const char* sources[1] = { source.data() };

	cl_int error = 0;
	cl_program program = clCreateProgramWithSource(context, 1, sources, lengths, &error);
	return program;
}


int main()
{
	int num_columns_a;
	int num_rows_a; 
	int num_columns_b; 
	int num_rows_b; 
	int* matrix_a; 
	int* matrix_b;
	int*matrix_output;
	int openCL_or_CPP;
	std::cout << "How many columns in the first matrix?\n";
	std::cin >> num_columns_a; 
	num_rows_b = num_columns_a; 
	std::cout << "Since there must be as many rows in the second matrix as there are columns in the first, there are " << num_rows_b << " rows in the second matrix\n";
	std::cout << "How many rows in the first matrix?\n";
	std::cin >> num_rows_a; 
	std::cout << "How many columns in the second matrix?\n";
	std::cin >> num_columns_b;
	int min = std::min(std::min(num_rows_a,num_columns_a), std::min(num_rows_b, num_columns_b));
	if (min <= 0)
	{
		std::cout << "Invalid input; all values must be above 0\n";
		return -1;
	}
	matrix_a = new int[num_rows_a * num_columns_a];
	for (int i = 0; i < num_rows_a; i++)
	{
		for (int j = 0; j < num_columns_a; j++)
		{
			matrix_a[(i * num_columns_a) + j] = i + j * i + j;
		}
	}
	matrix_b = new int[num_rows_b * num_columns_b];
	for (int i = 0; i < num_rows_b; i++)
	{
		for (int j = 0; j < num_columns_b; j++)
		{
			matrix_b[(i * num_columns_b) + j] = j * 3 + 1 + i;
		}
	}
	std::cout << "Do you want to multiply these matrices using OpenCl or regular C++?(0 or 1)\n";
	std::cin >> openCL_or_CPP;
	if (openCL_or_CPP == 0)
	{

	}

	else
	{
	printMatrix(matrix_a, num_columns_a, num_rows_a);
	printMatrix(matrix_b, num_columns_b, num_rows_b);
	 matrix_output = multiplyMatrices(matrix_a, matrix_b, num_columns_a, num_rows_a, num_columns_b, num_rows_b);
	  printMatrix(matrix_output, num_columns_b, num_rows_a);
	}
	return 0; 
}


int*multiplyMatricesOpenCL(int* matrix_a, int* matrix_b, int columns_a, int rows_a, int columns_b, int rows_b)
{
	OpenCLContext *context = new OpenCLContext();
	cl_int error = 0;
	cl_mem buf_matrix_a = clCreateBuffer(context->GetClContext(), CL_MEM_READ_ONLY | CL_MEM_USE_HOST_PTR, sizeof(int) * columns_a * rows_a, matrix_a, &error);
	cl_mem buf_matrix_b = clCreateBuffer(context->GetClContext(), CL_MEM_READ_ONLY | CL_MEM_USE_HOST_PTR, sizeof(int) * columns_b * rows_b, matrix_b, &error);
	cl_mem out_buffer = clCreateBuffer(context->GetClContext(), CL_MEM_WRITE_ONLY, sizeof(int) * rows_a * columns_b, nullptr, &error);
	cl_program program = CreateProgram("multiply.cl", context->GetClContext());
	cl_kernel kernel = clCreateKernel(program, "MULITPLY", &error);
	clSetKernelArg(kernel, 0, sizeof(cl_mem), buf_matrix_a);
	clSetKernelArg(kernel, 1, sizeof(cl_mem), buf_matrix_b);
	clSetKernelArg(kernel, 2, sizeof(cl_mem), out_buffer);
}

int* multiplyMatrices(int* matrix_a, int* matrix_b, int columns_a,int rows_a, int columns_b, int rows_b)
{
	int* matrix_output;
	int columns_output, rows_output;
	int temp;
	rows_output = rows_a;
	columns_output = columns_b;
	matrix_output = new int[rows_output * columns_output];
	for (int i = 0; i < rows_output; i++)
	{
		for (int j = 0; j < columns_output; j++)
		{
			temp = 0;
			for (int k = 0; k < columns_a; k++)
			{
				int matrix_a_result = matrix_a[(columns_a * i) + k];
				int matrix_b_result =  matrix_b[(k * columns_b) + j];
				temp += matrix_a_result * matrix_b_result;
			}
			matrix_output[(i * columns_output) + j] = temp;
		}
	}
	return matrix_output;
}

void printMatrix(int *matrix, int columns, int rows)
{
	if (columns > 10000 | rows > 10000)
	{
		return;
	}
	std::cout << "Printing a matrix\n";
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < columns; j++)
		{
			std::cout << matrix[(i * columns) + j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "Done printing a matrix\n";
}